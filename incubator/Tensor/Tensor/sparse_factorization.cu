#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__device__ float getTensorElement(int *T_i, float *T_d, int l_i, int l_d, int i, int t, int q){
	int start = T_i[i];
	int end = ((i+1) < l_i) ? T_i[i+1] : l_d;
	float res = 0.0f;
	for(int i=start+2;i<end;i+=3){
		if(T_d[i-2] > t) break;
		else if(T_d[i-2] == t && T_d[i-1] == q){
			res = T_d[i];
			break;
		}
	}
	return res;
}

__global__ void factorAKernel ( float *T_i, float *T_d, int l_i, int l_d ,float *A, float *B, float *C, float *A_n, int n, int k)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	float sum_n = 0.1f, sum_d = 0.1f;
	float temp = 0.0f, ratio = 0.0f, error = 0.0f;
	
	int start = T_i[i];
	int end = ((i+1) < l_i) ? T_i[i+1] : l_d;
	int t,q;

	for(int ind=start+2; ind<end; ind+=3){
		t = T_d[ind-2];
		q = T_d[ind-1];

		error = 0.0f;
		for(int j=0;j<k;j++){
			error += A[i*k+j]*B[t*k+j]*C[q*k+j];
		}

		temp = B[t*k+j]*C[q*k+j];
		sum_n += temp * T_d[ind] / error;
		sum_d += temp;
	}
	
	A_n[i*k+j] = A[i*k+j]*(sum_n/sum_d);
}

__global__ void factorBKernel ( float *T_t, float *T_d, int l_t, int l_d, float *A, float *B, float *C, float *B_n, int n, int k){
	int t = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	float sum_n = 0.1f, sum_d = 0.1f;
	float temp = 0.0f, ratio = 0.0f, error = 0.0f;
	
	int start = T_t[t];
	int end = ((t+1) < l_t) ? T_t[t+1] : l_d;
	int i,q;

	for(int ind=start+2; ind<end; ind+=3){
		q = T_d[ind-2];
		i = T_d[ind-1];

		error = 0.0f;
		for(int j=0;j<k;j++){
			error += A[i*k+j]*B[t*k+j]*C[q*k+j];
		}

		temp = A[i*k+j]*C[q*k+j];
		sum_n += temp * T_d[ind] / error;
		sum_d += temp;
	}

	B_n[t*k+j] = B[t*k+j]*(sum_n/sum_d);
}

__global__ void factorCKernel ( float *T_q, float *T_d, int l_q, int l_d, float *A, float *B, float *C, float *C_n, int n, int k){
	int q = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	float sum_n = 0.1f, sum_d = 0.1f;
	float temp = 0.0f, ratio = 0.0f, error = 0.0f;
	
	int start = T_q[q];
	int end = ((q+1) < l_q) ? T_q[q+1] : l_d;
	int i,t;

	for(int ind=start+2; ind<end; ind+=3){
		i = T_d[ind-2];
		t = T_d[ind-1];

		error = 0.0f;
		for(int j=0;j<k;j++){
			error += A[i*k+j]*B[t*k+j]*C[q*k+j];
		}

		temp = A[i*k+j]*B[t*k+j];
		sum_n += temp * T_d[ind] / error;
		sum_d += temp;
	}

	C_n[q*k+j] = C[q*k+j]*sum_n/sum_d;
}

int main ( int argc, char *  argv [] )
{
	int n,k;
	 
	// Tensor loading
	scanf("%d",&n);
	scanf("%d",&k);
	int size = n*n*n;
    int numBytesT = size * sizeof ( float );
	int numBytesABC = (n*k) * sizeof(float);

	
	float * A = new float [n*k];
	float * B = new float [n*k];
	float * C = new float [n*k];
	
	for(int i=0;i<(n*k);i++){
		A[i] = (float)(rand()%10000) + 1.0f;
		B[i] = (float)(rand()%10000) + 1.0f;
		C[i] = (float)(rand()%10000) + 1.0f;
	}

	float * T = new float [size];
	float *Q = new float[size];
	
    for ( int i = 0; i < size; i++ ){
        T[i] = (float)(rand()%10000) +1.0f;
		Q[i] = T[i];
	}


    float *T_c = NULL;
	hipMalloc ( (void**)&T_c, numBytesT );
	
	float *A_cuda = NULL, *B_cuda = NULL, *C_cuda = NULL, *A_next_cuda = NULL, *B_next_cuda = NULL, *C_next_cuda = NULL;
	hipMalloc ( (void**)&A_cuda, numBytesABC );
	hipMalloc ( (void**)&B_cuda, numBytesABC );
	hipMalloc ( (void**)&C_cuda, numBytesABC );
	hipMalloc ( (void**)&A_next_cuda, numBytesABC );
	hipMalloc ( (void**)&B_next_cuda, numBytesABC );
	hipMalloc ( (void**)&C_next_cuda, numBytesABC );

    dim3 threads = dim3(k, n);
    dim3 blocks  = dim3(1, 1);

    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate ( &start );
    hipEventCreate ( &stop );
    hipEventRecord ( start, 0 );
    
	hipDeviceSynchronize();
	hipMemcpy      ( T_c, T, numBytesT, hipMemcpyHostToDevice );
	hipMemcpy      ( Q_c, Q, numBytesT, hipMemcpyHostToDevice );
	
	hipMemcpy      ( A_cuda, A, numBytesABC, hipMemcpyHostToDevice );
	hipMemcpy      ( B_cuda, B, numBytesABC, hipMemcpyHostToDevice );
	hipMemcpy      ( C_cuda, C, numBytesABC, hipMemcpyHostToDevice );

	
	bool flag = true;

	int l_i, l_d;

	for(int i=0;i<1000;i++){
		if(flag){	
			hipDeviceSynchronize();
			factorAKernel<<<blocks, threads>>>(Ti_ind_cuda, Ti_data_cuda, l_i, l_d_i, A_cuda, B_cuda, C_cuda, A_next_cuda,n,k);

			hipDeviceSynchronize();
			factorBKernel<<<blocks, threads>>>(Tt_ind_cuda, Tt_data_cuda, l_t, l_d_t, A_next_cuda, B_cuda, C_cuda, B_next_cuda, n,k);

			hipDeviceSynchronize();
			factorCKernel<<<blocks, threads>>>(Tq_ind_cuda, Tq_data_cuda, l_q, l_d_q, A_next_cuda, B_next_cuda,C_cuda,C_next_cuda,n,k);

		}
		else{
			hipDeviceSynchronize();
			factorAKernel<<<blocks, threads>>>(T_c,Q_c,A_next_cuda,B_next_cuda,C_next_cuda,A_cuda,n,k);
			
			hipDeviceSynchronize();
			factorBKernel<<<blocks, threads>>>(T_c,Q_c,A_cuda,B_next_cuda,C_next_cuda,B_cuda,n,k);
			
			hipDeviceSynchronize();
			factorCKernel<<<blocks, threads>>>(T_c,Q_c,A_cuda,B_cuda,C_next_cuda,C_cuda,n,k);
		}
		flag = !flag;
    
	}
	
	hipDeviceSynchronize();
	if(flag == false){
		hipMemcpy      ( A, A_next_cuda, numBytesABC, hipMemcpyDeviceToHost );
		hipMemcpy      ( B, B_next_cuda, numBytesABC, hipMemcpyDeviceToHost );
		hipMemcpy      ( C, C_next_cuda, numBytesABC, hipMemcpyDeviceToHost );
	}
	else{
		hipMemcpy      ( A, A_cuda, numBytesABC, hipMemcpyDeviceToHost );
		hipMemcpy      ( B, B_cuda, numBytesABC, hipMemcpyDeviceToHost );
		hipMemcpy      ( C, C_cuda, numBytesABC, hipMemcpyDeviceToHost );
	}
	hipMemcpy      ( Q, Q_c, numBytesT, hipMemcpyDeviceToHost );

	
	hipEventRecord ( stop, 0 );
    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &gpuTime, start, stop );

    printf("\ntime spent executing by the GPU: %.2f millseconds\n", gpuTime );
    
	printf("Matrix A\n");
	//for ( int i = 0; i < (n*k); i++ ) printf ( "%f ", A[i] );
	printf("\n");

	printf("Matrix B\n");
	//for ( int i = 0; i < (n*k); i++ ) printf ( "%f ", B[i] );
	printf("\n");

	printf("Matrix C\n");
	//for ( int i = 0; i < (n*k); i++ ) printf ( "%f ", C[i] );
	printf("\n");
	
	printf("Tensor Q\n");
	//for(int i=0;i<(n*n*n);i++) printf("%f ", Q[i]);
	printf("\n");

    hipEventDestroy ( start );
    hipEventDestroy ( stop  );
    hipFree(T_c);
	hipFree(Q_c);
	hipFree(A_cuda);
	hipFree(B_cuda);
	hipFree(C_cuda);
	hipFree(A_next_cuda);
	hipFree(B_next_cuda);
	hipFree(C_next_cuda);
    delete[] T;
	delete[] A;
	delete[] B;
	delete[] C;
	delete[] Q;

    return 0;
}